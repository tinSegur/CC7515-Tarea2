#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void sim_life(int n, int m, char *a, char *b){
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint j = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i < n && j < m) {
        int i0 = (i + n - 1)%n;
        int i2 = (i + 1)%n;

        int j0 = (j + m - 1)%m;
        int j2 = (j + 1)%m;

        char liveNeighbors =
                a[i0*m + j0] + a[i0*m + j] + a[i0*m + j2] +
                a[i*m + j0]  + a[i*m + j2] +
                a[i2*m + j0] + a[i2*m + j] + a[i2*m + j2];

        __syncthreads();

        b[i*m + j] = (liveNeighbors == 3) || (liveNeighbors == 2 && a[i*m + j]) ? 1 : 0;
    }

}

